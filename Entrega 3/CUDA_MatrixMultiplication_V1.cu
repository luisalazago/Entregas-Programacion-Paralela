#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16
#define MAT_COLS N
#define MAT_ROWS N
#define TILES 4

#define MAX_THREADS 512
#define BLOCKS 4
#define THREADS_PER_BLOCK 32

/***( CUDA kernel )*******************************************************/

__global__ void MatMulKernel(int *MA, int *MB, int *MY)
{
    // Initialise the shared matrix.
    __shared__ int shared_matrix[THREADS_PER_BLOCK][THREADS_PER_BLOCK];

    // Calculate the rows and cols of the matrix.
    int row = blockIdx.y * TILES + threadIdx.y;
    int col = blockIdx.x * TILES + threadIdx.x;

    int val = 0;

    // Prevent in excess threads to perform out-of-bounds operations.
    if (row < N && col < N)
    {
        for (int i = 0; i < N; i++)
            val += MA[row * N + i] * MB[i * N + col]; // Multiply the rows of the first matrix and the columns of the second matrix.
    }

    // Insert into the shared memory the product calculated before.
    shared_matrix[threadIdx.y][threadIdx.x] = val;
    __syncthreads();

    // We verify the size of the rows and columns exist in the matrix.
    if(row < N && col < N) {
        // In the final matrix we insert the values of the shared memory, this optimize the efficiency of the data pass.
        MY[((blockIdx.y * TILES + threadIdx.y) * N) + (blockIdx.x * TILES) + threadIdx.x] = shared_matrix[threadIdx.y][threadIdx.x];
    }
}

/*--( Support functions )------------------------------------------------*/

/* Initialise matrices of the given size */
void llenar(int *matrix, int fil, int col, bool a){
    /*
    Functions that initiallize the matrix that will be used to
    calculate the dot product
    Inputs:
        Matrix: is a matrix of FILxCOL
    Outputs
        Matrix: is a matrix of FILxCOL

    */
    int i, j;
    if(a) {
        for(i = 0; i < fil; ++i){
            for(j = 0; j < col; ++j) {
              matrix[(i*col)+j] = 1;
            }
        }
    }
    else {
        for(i = 0; i < fil; ++i){
            for(j = 0; j < col; ++j) {
              matrix[(i*col)+j] = 2;
            }
        }
    }
}

/* Print matrix of the given size */
void PrintMatrix(int *matrix)
{
    int row,
        col;

    for (row = 0; row < N; row++)
    {
        for (col = 0; col < N; col++)
            printf("%7d ", matrix[row * N + col]);
        printf("\n");
    }
    printf("\n");
}

/*--( Main function )----------------------------------------------------*/

int main(void)
{
    int *MA,
        *MB,
        *MY;
    int *d_MA,
        *d_MB,
        *d_MY;

    int size = N * N * sizeof(int);

    MA = (int *)malloc(size);
    MB = (int *)malloc(size);
    MY = (int *)malloc(size);

    // Initialise A, B matrices
    llenar(MA, N, N, true);
    llenar(MB, N, N, false);

    printf("\n%dx%d A matrix is \n", N, N);
    PrintMatrix(MA);

    printf("\n%dx%d B matrix is \n", N, N);
    PrintMatrix(MB);

    // Initialise the Y (final) matrix.
    MY = (int *)malloc(size);
    memset((void *)MY, 0, size);

    // Allocate in the DRAM the space and data from the CPU to the GPU.
    hipMalloc(&d_MA, size);
    hipMalloc(&d_MB, size);

    hipMemcpy(d_MA, MA, size, hipMemcpyHostToDevice);
    hipMemcpy(d_MB, MB, size, hipMemcpyHostToDevice);

    hipMalloc(&d_MY, size);
    hipMemcpy(d_MY, MY, size, hipMemcpyHostToDevice);

    dim3 BlocksPerGrid(1, 1);
    dim3 ThreadsPerBlock(N, N);
    if (N * N > MAX_THREADS)
    {
        ThreadsPerBlock.x = MAX_THREADS;
        ThreadsPerBlock.y = MAX_THREADS;
        BlocksPerGrid.x = ceil((double)N / (double)ThreadsPerBlock.x);
        BlocksPerGrid.y = ceil((double)N / (double)ThreadsPerBlock.y);
    }

    MatMulKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(d_MA, d_MB, d_MY);

    // Get the data from the Y matrix filled in the kernel.
    hipMemcpy(MY, d_MY, size, hipMemcpyDeviceToHost);

    printf("\n%dx%d Y matrix is \n", N, N);
    PrintMatrix(MY);

    hipFree(d_MA);
    hipFree(d_MB);
    hipFree(d_MY);

    free(MA);
    free(MB);
    free(MY);
}
